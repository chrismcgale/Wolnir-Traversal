#include "hip/hip_runtime.h"
struct CSRGraph {
    int numVertices;
    unsigned int* srcPtrs;
    unsigned int* dst;
}


__host__ unsigned int* frontier_bfs(CSRGraph csrGraph, unsigned int startVertex, unsigned int max_degree) {
    CSRGraph* csrGraph_d;
    unsigned int* numCurrFrontier_h, *numCurrFrontier_d;
    *numCurrFrontier_h = 1

    unsigned int level_h[csrGraph.numVertices], *level_d;

    memset(level_h, UINT_MAX, sizeof(unsigned int) * csrGraph.numVertices);

    // Can have multiple starts!!
    level_h[startVertex] = 0;

    hipMalloc((void**)&csrGraph_d, sizeof(CSRGraph));
    hipMalloc((void**)&level_d, sizeof(level));
    hipMalloc((void**)&numCurrFrontier_d, sizeof(unsigned int));

    hipMemcpy(csrGraph_d, csrGraph, sizeof(CSRGraph) s, hipMemcpyHostToDevice);
    hipMemcpy(level_d, level_h, sizeof(unsigned int) * csrGraph.numVertices, hipMemcpyHostToDevice);
    hipMemcpy(numCurrFrontier_d, numCurrFrontier_h, sizeof(unsigned int), hipMemcpyHostToDevice);


    unsigned int numPrevFrontier = 1;
    unsigned int* prevFrontier_h, *prevFrontier_d;
    hipMalloc((void**)&prevFrontier_d, sizeof(unsigned int) * max_degree);

    while (numPrevFrontier > 0) {
        hipMemcpy(prevFrontier_d, prevFrontier_h, sizeof(unsigned int), hipMemcpyHostToDevice);

        unsigned int* currFrontier; 
        hipMalloc((void**)&currFrontier, sizeof(unsigned int) * max_degree);

        *numCurrFrontier_h = 0;
        hipMemcpy(numCurrFrontier_d, numCurrFrontier_h, sizeof(unsigned int), hipMemcpyHostToDevice);

        frontier_bfs_kernel<<<(csrGraph.numVertices / 256), 256>>>(csrGraph, level, newVertex, prevFrontier_d, numPrevFrontier, currFrontier, numCurrFrontier_d, currLevel);

        hipMemcpy(prevFrontier_h, currFrontier, sizeof(unsigned int) * max_degree, hipMemcpyDeviceToHost);
        hipMemcpy(numPrevFrontier, numCurrFrontier_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

        hipFree(currFrontier);
        currLevel++;
    }

    hipMemcpy(level_h, level_d, sizeof(unsigned int) * csrGraph.numVertices, hipMemcpyDeviceToHost);

    hipFree(csrGraph);
    hipFree(level_d);
    hipFree(numCurrFrontier_d);
    hipFree(prevFrontier);

    return level_h;

}




__global__ void frontier_bfs_kernel(CSRGraph csrGraph, unsigned int* level, 
                unsigned int* prevFrontier, unsigned int numPrevFrontier, 
                unsigned int* currFrontier. unsigned int* numCurrFrontier, unsigned int currLevel) {
    unsigned int i = blockIdx.x*blockDim.x + thread.Idx.x;
    if (i < numPrevFrontier) {
        unsigned int vertex = prevFrontier[i];
        for (unsigned int edge = csrGraph.srcPtrs[vertex]; edge < csrGraph.srcPtrs[vertex + 1]; edge++) {
            unsigned int neighbour = csrGraph.dst[edge];
            if (atomicCAS(&level[neighbour], UINT_MAX, currLevel) == UINT_MAX) { // Not yet visited
                unsigned int currFrontierIdx = atomicAdd(numCurrFrontier, 1);
                currFrontier[currFrontierIdx] = neighbour;
            }
        }
    }
}